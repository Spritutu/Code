#include "hip/hip_runtime.h"
#include "CudaParallel.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
//#define _USE_MATH_DEFINES
#include <cmath>
#include "Type.h"

//ʹ��GPU���� �˺���
__global__ void CudaCaculateKernal(double* pX,double* pY,int nLength)
{
	if (nullptr == pX || nullptr == pY)
	{
		return;
	}
	//blockDim �߳̿��е��߳���
	//gridDim �����е��߳̿���
	//threadidx �߳̿��е��߳�����
	//blockidx �߳̿��������е�����
	int nIndex = blockDim.x * blockIdx.x + threadIdx.x;


	if (nIndex < 0 || nIndex >= nLength)
	{
		return;
	}

	pY[nIndex] = sin(pX[nIndex]);
	printf("nIndex = %d.\n", nIndex);

}

//����Cuda���в��м���ĺ���
void CalculateSinValue(int nLength)
{
	//�������ڴ��е����ݿ������豸�ڴ���
	//�����й�����
	double* pCudaX = NULL;
	double* pCudaY = NULL;

	//�й�����(���������豸�˶����Է���)
	hipMallocManaged(&pCudaX, nLength * sizeof(double));
	hipMallocManaged(&pCudaY, nLength * sizeof(double));

	double dStep = 2.0 * PI / nLength;

	for (int i = 0; i < nLength; ++i)
	{
		pCudaX[i] = i * dStep;
	}
	//�������м���������
	int n = (nLength + BLOCK_THREAD_NUM - 1) / BLOCK_THREAD_NUM;
	auto BlockSize = dim3(n, 1, 1);
	uint3 ThreadSize;
	ThreadSize.x = BLOCK_THREAD_NUM;
	ThreadSize.y = 1;
	ThreadSize.z = 1;
	//�����˺���
	CudaCaculateKernal <<<BlockSize, ThreadSize>>> (pCudaX, pCudaY, nLength);
	//�ȴ����������е��߳̽���
	hipDeviceSynchronize();
	//�ͷ��ڴ�
	hipFree(pCudaX);
	hipFree(pCudaY);
}

//GPU����������ѯ   ����������1.0��ʼ
void QueryGPUCapcity(void)
{
	hipDeviceProp_t prop;

	int count = 0;
	hipGetDeviceCount(&count);
	printf("�Կ���֧�ֵ�cuda������������%d\n", count);
	for (int i = 0; i < count; ++i) 
	{
		hipGetDeviceProperties(&prop, i);
		printf("----��%d���������Ļ�����Ϣ----\n", i + 1);
		printf("���������ƣ�%s \n", prop.name);
		printf("����������%d.%d\n", prop.major, prop.minor);
		printf("�豸��ȫ���ڴ�������%dMB\n", prop.totalGlobalMem / 1024 / 1024);
		printf("�豸�ϳ����ڴ�������%dKB\n", prop.totalConstMem / 1024);
		printf("һ���߳̿��п�ʹ�õ�������ڴ棺%dKB\n", prop.sharedMemPerBlock / 1024);
		printf("һ���߳����������߳�������%d\n", prop.warpSize);
		printf("һ���߳̿��пɰ���������߳�������%d\n", prop.maxThreadsPerBlock);
		printf("��ά�߳̿�������ÿһά�ɰ���������߳�������(%d,%d,%d)\n", prop.maxThreadsDim[0],
			prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("һ���̸߳���ÿһά�ɰ���������߳̿�������(%d,%d,%d)\n", prop.maxGridSize[0],
			prop.maxGridSize[1], prop.maxGridSize[2]);
	}
}